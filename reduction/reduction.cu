#include "hip/hip_runtime.h"
#include <timeprinter/printer.hpp>
#include <util/to_scalar.hpp>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <thrust/count.h>
#include <thrust/reduce.h>
#include <thrust/extrema.h>

#include <cassert>
#include <random>
#include <vector>

namespace
{
    tp::printer g_tpr;

    namespace detail
    {
        template<typename T>
        struct int_distribution_traits
        {
            using value_type = T;
            using distribution = std::uniform_int_distribution<value_type>;
            static constexpr value_type lower = T{};
            static constexpr value_type upper = 100;
        };

        template<typename T>
        struct real_distribution_traits
        {
            using value_type = T;
            using distribution = std::uniform_real_distribution<value_type>;
            static constexpr value_type lower = T{};
            static constexpr value_type upper = 1.0;
        };
    };

    enum class placement
    {
        host,
        device,
    };

    enum class operation
    {
        sum,
        mult,
        min,
        max,
        count_if,
    };

    enum class element_type
    {
        int32,
        int64,
        real32,
        real64,
    };

    struct host_placement
    {
        template<typename T>
        using container = thrust::host_vector<T>;
        static const decltype(thrust::host) policy;
    };

    struct device_placement
    {
        template<typename T>
        using container = thrust::device_vector<T>;
        static const decltype(thrust::device) policy;
    };

    decltype(host_placement::policy) host_placement::policy = thrust::host;
    decltype(device_placement::policy) device_placement::policy = thrust::device;

    template<typename T>
    struct sum
    {
        using value_type = T;
        using functor = thrust::plus<value_type>;
        static constexpr value_type init_value = 0;
    };

    template<typename T>
    struct mult
    {
        using value_type = T;
        using functor = thrust::multiplies<value_type>;
        static constexpr value_type init_value = 1;
    };

    template<typename>
    struct distribution_traits {};

    template<>
    struct distribution_traits<std::int32_t> :
        detail::int_distribution_traits<std::int32_t>
    {};

    template<>
    struct distribution_traits<std::int64_t> :
        detail::int_distribution_traits<std::int64_t>
    {};

    template<>
    struct distribution_traits<float> :
        detail::real_distribution_traits<float>
    {};

    template<>
    struct distribution_traits<double> :
        detail::real_distribution_traits<double>
    {};

    template<typename T>
    struct element_traits
    {
        using value_type = T;
        using dist_traits = distribution_traits<value_type>;
        static constexpr value_type count_threshold =
            (dist_traits::lower + dist_traits::upper) / 2;
    };

    struct greater_than_threshold
    {
        __host__ __device__ bool operator()(const std::int32_t& x) { return call_impl(x); }
        __host__ __device__ bool operator()(const std::int64_t& x) { return call_impl(x); }
        __host__ __device__ bool operator()(const float& x) { return call_impl(x); }
        __host__ __device__ bool operator()(const double& x) { return call_impl(x); }

    private:
        template<typename T>
        __host__ __device__
            bool call_impl(const T& x)
        {
            return x >= element_traits<T>::count_threshold;
        }
    };

    struct arguments
    {
        placement where = placement::host;
        operation op = operation::sum;
        element_type etype = element_type::int32;
        std::size_t n = 0;
        std::size_t iters = 0;

        arguments(int argc, const char* const* argv)
        {
            if (argc < 6)
            {
                print_usage(argv[0]);
                throw std::invalid_argument("Not enough arguments");
            }
            where = get_placement(argv[1]);
            op = get_operation(argv[2]);
            etype = get_element_type(argv[3]);
            util::to_scalar(argv[4], n);
            assert(n > 0);
            if (!n)
                throw std::invalid_argument("n must be greater than 0");
            util::to_scalar(argv[5], iters);
            assert(iters > 0);
            if (!iters)
                throw std::invalid_argument("iters must be greater than 0");
        }

    private:
        std::string lowercase(const char* str)
        {
            std::string lower(str);
            auto tolower = [](unsigned char c) { return std::tolower(c); };
            for (char& c : lower)
                c = tolower(c);
            return lower;
        }

        placement get_placement(const char* arg)
        {
            std::string lower = lowercase(arg);
            if (lower == "host")
                return placement::host;
            if (lower == "device")
                return placement::device;
            throw std::invalid_argument("invalid placement");
        }

        operation get_operation(const char* arg)
        {
            std::string lower = lowercase(arg);
            if (lower == "sum")
                return operation::sum;
            if (lower == "mult")
                return operation::mult;
            if (lower == "min")
                return operation::min;
            if (lower == "max")
                return operation::max;
            if (lower == "count_if")
                return operation::count_if;
            throw std::invalid_argument("invalid operation");
        }

        element_type get_element_type(const char* arg)
        {
            std::string lower = lowercase(arg);
            if (lower == "i32")
                return element_type::int32;
            if (lower == "i64")
                return element_type::int64;
            if (lower == "r32")
                return element_type::real32;
            if (lower == "r64")
                return element_type::real64;
            throw std::invalid_argument("invalid element type");
        }

        void print_usage(const char* prog)
        {
            std::cerr << "Usage: " << prog
                << " <host|device> <operation> <type> <n> <iters>\n";
            std::cerr << "\toperation: sum | mult | min | max | count_if\n";
            std::cerr << "\ttype: i32 | i64 | r32 | r64\n";
        }
    };

    template<typename T>
    __attribute__((noinline)) thrust::host_vector<T> random_vector(
        std::size_t n,
        std::mt19937_64& engine)
    {
        using traits = element_traits<T>;
        tp::sampler smp(g_tpr);
        (void)smp;
        typename traits::dist_traits::distribution dist{
            traits::dist_traits::lower, traits::dist_traits::upper
        };
        thrust::host_vector<typename traits::value_type> vec(n);
        for (typename traits::value_type& val : vec)
            val = dist(engine);
        return vec;
    }

    template<typename Placement, typename BinaryOp>
    __attribute__((noinline)) typename BinaryOp::value_type reduce_work(
        const typename Placement::container<typename BinaryOp::value_type>& vec,
        std::size_t iters)
    {
        using value_type = typename BinaryOp::value_type;
        tp::sampler smp(g_tpr);
        (void)smp;
        thrust::host_vector<value_type> results(iters);
        for (std::size_t i = 0; i < iters; i++)
            results[i] = thrust::reduce(
                Placement::policy,
                std::begin(vec),
                std::end(vec),
                BinaryOp::init_value,
                typename BinaryOp::functor{});
        return results.front();
    }

    template<typename Placement, typename T>
    __attribute__((noinline)) T count_if_work(
        const typename Placement::container<T>& vec,
        std::size_t iters)
    {
        tp::sampler smp(g_tpr);
        (void)smp;
        thrust::host_vector<T> results(iters);
        for (std::size_t i = 0; i < iters; i++)
            results[i] = thrust::count_if(
                Placement::policy,
                std::begin(vec),
                std::end(vec),
                greater_than_threshold{});
        return results.front();
    }

    template<typename Placement, typename T>
    __attribute__((noinline)) T max_element_work(
        const typename Placement::container<T>& vec,
        std::size_t iters)
    {
        tp::sampler smp(g_tpr);
        (void)smp;
        thrust::host_vector<T> results(iters);
        for (std::size_t i = 0; i < iters; i++)
            results[i] = *thrust::max_element(
                Placement::policy, std::begin(vec), std::end(vec));
        return results.front();
    }

    template<typename Placement, typename T>
    __attribute__((noinline)) T min_element_work(
        const typename Placement::container<T>& vec,
        std::size_t iters)
    {
        tp::sampler smp(g_tpr);
        (void)smp;
        thrust::host_vector<T> results(iters);
        for (std::size_t i = 0; i < iters; i++)
            results[i] = *thrust::min_element(
                Placement::policy, std::begin(vec), std::end(vec));
        return results.front();
    }

    template<typename BinaryOp>
    __attribute__((noinline)) typename BinaryOp::value_type reduce_host(
        std::size_t n,
        std::size_t iters,
        std::mt19937_64& engine)
    {
        auto vec = random_vector<typename BinaryOp::value_type>(n, engine);
        return reduce_work<host_placement, BinaryOp>(vec, iters);
    }

    template<typename BinaryOp>
    __attribute__((noinline)) typename BinaryOp::value_type reduce_device(
        std::size_t n,
        std::size_t iters,
        std::mt19937_64& engine)
    {
        auto vec = random_vector<typename BinaryOp::value_type>(n, engine);
        thrust::device_vector<typename BinaryOp::value_type> d_vec = vec;
        return reduce_work<device_placement, BinaryOp>(d_vec, iters);
    }

    template<typename T>
    __attribute__((noinline)) T count_if_host(
        std::size_t n,
        std::size_t iters,
        std::mt19937_64& engine)
    {
        auto vec = random_vector<T>(n, engine);
        return count_if_work<host_placement>(vec, iters);
    }

    template<typename T>
    __attribute__((noinline)) T count_if_device(
        std::size_t n,
        std::size_t iters,
        std::mt19937_64& engine)
    {
        auto vec = random_vector<T>(n, engine);
        thrust::device_vector<T> d_vec = vec;
        return count_if_work<device_placement>(d_vec, iters);
    }

    template<typename T>
    __attribute__((noinline)) T max_element_host(
        std::size_t n,
        std::size_t iters,
        std::mt19937_64& engine)
    {
        auto vec = random_vector<T>(n, engine);
        return max_element_work<host_placement>(vec, iters);
    }

    template<typename T>
    __attribute__((noinline)) T max_element_device(
        std::size_t n,
        std::size_t iters,
        std::mt19937_64& engine)
    {
        auto vec = random_vector<T>(n, engine);
        thrust::device_vector<T> d_vec = vec;
        return max_element_work<device_placement>(d_vec, iters);
    }

    template<typename T>
    __attribute__((noinline)) T min_element_host(
        std::size_t n,
        std::size_t iters,
        std::mt19937_64& engine)
    {
        auto vec = random_vector<T>(n, engine);
        return min_element_work<host_placement>(vec, iters);
    }

    template<typename T>
    __attribute__((noinline)) T min_element_device(
        std::size_t n,
        std::size_t iters,
        std::mt19937_64& engine)
    {
        auto vec = random_vector<T>(n, engine);
        thrust::device_vector<T> d_vec = vec;
        return min_element_work<device_placement>(d_vec, iters);
    }

    void dispatch_work(const arguments& args, std::mt19937_64& engine)
    {
    #define DISPATCH_ALL_TYPES(macro) \
        macro(int32); \
        macro(int64); \
        macro(real32); \
        macro(real64)

    #define DISPATCH_REDUCE(place, type, op_type) \
        do { \
            if (args.where == placement::place && \
                args.etype == element_type::type && \
                args.op == operation::op_type) \
            { \
                std::cerr << #place " " #type " " #op_type "\n"; \
                std::cerr << reduce_ ##place<op_type<type ## _t>>(args.n, args.iters, engine) << "\n"; \
                return; \
            } \
        } while (false)

    #define DISPATCH_REDUCE_ALL(type) \
        DISPATCH_REDUCE(host, type, sum); \
        DISPATCH_REDUCE(host, type, mult); \
        DISPATCH_REDUCE(device, type, sum); \
        DISPATCH_REDUCE(device, type, mult)

    #define DISPATCH_COUNT_IF(place, type) \
        do { \
            if (args.where == placement::place && \
                args.etype == element_type::type && \
                args.op == operation::count_if) \
            { \
                std::cerr << #place " " #type " count_if\n"; \
                std::cerr << count_if_ ##place<type ## _t>(args.n, args.iters, engine) << "\n"; \
                return; \
            } \
        } while (false)

    #define DISPATCH_COUNT_IF_ALL(type) \
        DISPATCH_COUNT_IF(host, type); \
        DISPATCH_COUNT_IF(device, type)

    #define DISPATCH_EXTREMA(place, type, op_type) \
        do { \
            if (args.where == placement::place && \
                args.etype == element_type::type && \
                args.op == operation::op_type) \
            { \
                std::cerr << #place " " #type " " #op_type "\n"; \
                std::cerr << op_type ## _element_ ##place<type ## _t>(args.n, args.iters, engine) \
                    << "\n"; \
                return; \
            } \
        } while (false)

    #define DISPATCH_EXTREMA_ALL(type) \
        DISPATCH_EXTREMA(host, type, max); \
        DISPATCH_EXTREMA(host, type, min); \
        DISPATCH_EXTREMA(device, type, max); \
        DISPATCH_EXTREMA(device, type, min)

        using real32_t = float;
        using real64_t = double;
        DISPATCH_ALL_TYPES(DISPATCH_REDUCE_ALL);
        DISPATCH_ALL_TYPES(DISPATCH_COUNT_IF_ALL);
        DISPATCH_ALL_TYPES(DISPATCH_EXTREMA_ALL);
        throw std::runtime_error("unable to dispatch work");
    }
}

int main(int argc, char** argv)
{
    try
    {
        const arguments args{ argc, argv };
        std::random_device rnd_dev;
        std::mt19937_64 engine{ rnd_dev() };
        dispatch_work(args, engine);
    }
    catch (const std::exception& e)
    {
        std::cerr << e.what() << '\n';
        return 1;
    }
}
